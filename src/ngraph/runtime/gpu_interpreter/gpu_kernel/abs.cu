#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// Copyright 2017 Nervana Systems Inc.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// ----------------------------------------------------------------------------

#ifndef _CUDA_VEC_ABS_
#define _CUDA_VEC_ABS_

#include "abs.hpp"
#include <hip/hip_runtime.h>

__global__ void VecAbs(float* A, float* B) 
{ 
    int i = threadIdx.x; 
    B[i] = A[i] < 0 ? -A[i] : A[i]; 
} 

extern "C"
void runVecAbs(float* arg, float* out, size_t count)
{
	float *d_arg, *d_out;
	
	hipMalloc((void **)& d_arg, sizeof(float) * count);
	hipMalloc((void **)& d_out, sizeof(float) * count);
  	
	hipMemcpy(d_arg, arg, count, hipMemcpyHostToDevice);
	
        VecAbs<<<1, count>>>(d_arg, d_out);

	hipMemcpy(out, d_out, count, hipMemcpyDeviceToHost);
	
	hipFree(d_arg);
	hipFree(d_out);
}
#endif
